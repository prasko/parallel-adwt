/*
  Copyright: (c) 2011 Matija Osrecki <matija.osrecki@fer.hr>
*/

#include <vector>

#include "combine.h"

// Floating point type to be used on the device.
typedef float dftype; 

void CUDACombiner::combine(std::vector<double> &res) {
  const int n = (int)x_.size();
  const int typesize = sizeof(dftype);

  // calculate Xt * X and Xt * Y
  dftype *hxx = new dftype[n+1];
  dftype *hxy = new dftype[n+1];

  hxx[0] = hxy[0] = 0.0;

  for(int i = 0; i < n; ++i) {
    hxx[i+1] = hxx[i] + x_[i] * x_[i];
    hxy[i+1] = hxy[i] + x_[i] * y_[i];
  }

  // allocate device memory and copy data

  dftype *dxx, *dxy, *dres;

  hipMalloc(&dxx, (n+1) * typesize);
  hipMalloc(&dxy, (n+1) * typesize);
  hipMalloc(&dres, n * typesize);

  hipMemcpy(dxx, hxx, (n+1) * typesize, hipMemcpyHostToDevice);
  hipMemcpy(dxy, hxy, (n+1) * typesize, hipMemcpyHostToDevice);

  // TODO(matija): init kernel

  // TODO(matija): start kernel

  // retrieve results
  dftype *hres = new dftype[n];
  hipMemcpy(hres, dres, n * typesize, hipMemcpyDeviceToHost);

  res.resize(n);
  copy(hres, hres+n, res, res+n);

  // free memory
  delete[] hxx;
  delete[] hxy;
  delete[] hres;

  hipFree(dxx);
  hipFree(dxy);
  hipFree(dres);
}

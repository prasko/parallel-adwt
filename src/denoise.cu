#include "hip/hip_runtime.h"
/*
  Copyright: (c) 2011 Matija Osrecki <matija.osrecki@fer.hr>
*/

#include <cassert>
#include <algorithm>
#include <vector>

#include "denoise.h"

typedef float Decimal;

using std::vector;

__global__ void denoise_kernel(const Decimal *signals, 
                               Decimal *param, 
                               Decimal *wsize,
                               const int n,
                               const int m,
                               const Denoise::CUDAICIDenoiser denoiser) {

  // init starting thread responsibility
  int signal_pos = blockIdx.x * blockDim.x * thread_jobs + threadIdx.x;
  const int signal_index = blockIdx.y * blockDim.y + threadIdx.y;

  if(signal_index >= n) return;

  // init variables
  Decimal sum, avg, tavg, curr_sigma, rk;
  Decimal maxlb, minub;

  // window size
  int wsize;

  for(int job = 0; job < thread_jobs; ++job) {
    if(signal_pos >= m) return;
    
    // init max upper and min lower bounds
    maxlb = -1e7;
    minub = +1e7;

    // increment wsize - window size
    for(wsize = 2; signal_pos+wsize-1 < m; ++wsize) {
      // calculate current sum, temp. average and sigma
      sum += signals[signal_index*m+signal_pos+wsize-1];
      tavg = sum / wsize;
      curr_sigma = denoiser.sigma / sqrt(wsize);

      // recalculate max upper and min lower bounds
      minub = std::min(minub, tavg + denoiser.gama * curr_sigma);
      maxlb = std::max(maxlb, tavg - denoiser.gama * curr_sigma);

      // calculate new RICI parameter Rk
      rk = (minub - maxlb) / (2 * gama * curr_sigma);

      // break if ICI or RICI conditions achieved
      if(minub < maxlb || rk < denoiser.rc) break;

      avg = tavg;
    }

    // add average and window size to solution
    param[signal_index*m+signal_pos] = avg;
    wsize[signal_index*m+signal_pos] = wsize-1;

    signal_pos += blockDim.x;
  }
}

namespace Denoise {

  Denoiser::Result* CUDAICIDenoiser::denoise(const Signal &sig) {
    std::vector<Signal> multiple_sig;
    std::vector<Denoiser::Result*> multiple_res;

    multiple_sig.push_back(sig);
    denoiseMultiple(multiple_sig, multiple_res);
    return multiple.res[0];
  }

  void CUDAICIDenoiser::denoiseMultiple(const vector<Signal> &sig,
                                        vector<Denoiser::Result*> &res) {

    assert(res.empty());

    if(sig.empty()) {
      return;
    }

    // 1. init number of signals, signal width, and width in bytes

    const int n = (int)sig.size();
    int m = (int)sig[0].size();
    bool same_width = true;

    for(int i = 1; i < n; ++i) {
      if((int)sig[i].size() > m) {
        same_width = false;
        m = sig[i];
      }
    }

    const int width_bytes = m*sizeof(Decimal);
    const int size_bytes = 2*n*width_bytes;

    // 2. init input data on host and device

    Decimal *host_signals = new Decimal[2*n*m];

    // set all to 0 if they're not the same length
    if(!same_width) {
      memset(host_signals, 0, sizeof host_signals);
      // XXX maybe use forloop instead!!
      // for(int i = 0; i < 2*n*m; ++i) 
      //   host_signals[i] = 0.0;
    }

    for(int i = 0; i < n; ++i) {
      std::copy(sig[i].begin(), sig[i].end(), 
                host_signals+2*i*width);

      std::reverse_copy(sig[i].begin(), sig[i].end(), 
                        host_signals+(2*i+2)*m-(int)sig[i].size());
    }

    Decimal *device_signals;
    hipMalloc(&device_signals, size_bytes);
    hipMemcpy(device_signals, host_signals, size_bytes, 
               hipMemcpyHostToDevice);

    // 2. init result data

    Decimal *device_param;
    int *device_wsize;

    hipMalloc(&device_param, size_bytes);
    hipMalloc(&device_param, 2*n*sizeof(int));

    // 3. init kernel

    const int block_size = 16;
    const int thread_spread = 32;

    const int grid_width = (n-1) / (block_size_ * thread_jobs_) + 1;
    const int grid_height = (n-1) / block_size_ + 1;

    dim3 dim_block(block_size, block_size);
    dim3 dim_grid(grid_width, grid_height);

    // 4. start kernel

    denoise_kernel<<<dim_grid, dim_block>>>(device_signals, 
                                            device_param,
                                            device_wsize, 
                                            n,
                                            m,
                                            *this);

    // 5. retrieve and post-calculate results

    Decimal *host_param = new Decimal[2*n*m];
    int *host_wsize = new int[2*n*m];

    hipMemcpy(host_param, device_param, size_bytes, 
               hipMemcpyDeviceToHost);

    hipMemcpy(host_wsize, device_wsize, 2*n*sizeof(int), 
               hipMemcpyDeviceToHost);

    res.resize(n);
    for(int i = 0; i < n; ++i) {
      Decimal pvleft, pvright;  // param value
      int wsleft, wsright;  // window size

      for(int pos = 0; pos < (int)sig[i].size(); ++pos) {
        pvright = host_param[2*i*m + pos];
        pvleft = host_param[(2*i+2)*m - (pos+1)]; // (2*i+1)*m + (m-pos-1)
        wsright = host_wsize[2*i*m + pos];
        wsleft = host_wsize[(2*i+2)*m - (pos+1)];
        
        res.back()->addLeft(wsleft);
        res.back()->addRight(wsright);
        res.back()->addDenoised((pvright * wsright + pvleft * wsleft) /
                                (wsleft + wsright));
      }
    }

    // 6. clean up

    delete[] host_signals;
    delete[] host_param;
    delete[] host_wsize;

    hipFree(device_signals);
    hipFree(device_param);
    hipFree(device_wsize);
  }
                                      
}  // namespace

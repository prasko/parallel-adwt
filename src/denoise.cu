/*
  Copyright: (c) 2011 Matija Osrecki <matija.osrecki@fer.hr>
*/

#include <cassert>
#include <algorithm>
#include <vector>

#include "denoise.h"

typedef float Decimal;

namespace Denoise {

  Denoiser::Result* CUDAICIDenoiser::denoise(const Signal &sig) {
    std::vector<Signal> multiple_sig;
    std::vector<Denoiser::Result*> multiple_res;

    multiple_sig.push_back(sig);
    denoiseMultiple(multiple_sig, multiple_res);
    return multiple.res[0];
  }

  void CUDAICIDenoiser::denoiseMultiple(const std::vector<Signal> &sig,
      std::vector<Denoiser::Result*> &res) {

    if(sig.empty()) return;

    // 1. init number of signals, signal width, and width in bytes

    const int n = (int)sig.size();
    int m = (int)sig[0].size();
    bool same_width = true;

    for(int i = 1; i < n; ++i)
      if((int)sig[i].size() > m) {
        same_width = false;
        m = sig[i];
      }
  
    const int width_bytes = m*sizeof(Decimal);
    const int size_bytes = 2*n*width_bytes;

    // 2. init input data on host and device

    Decimal *host_signals = new Decimal[2*n*m];

    // set all to 0 if they're not the same length
    if(!same_width) {
      memset(host_signals, 0, sizeof host_signals);
      // XXX maybe use forloop instead!!
      // for(int i = 0; i < 2*n*m; ++i) 
      //   host_signals[i] = 0.0;
    }

    for(int i = 0; i < n; ++i) {
      std::copy(sig[i].begin(), sig[i].end(), 
                host_signals+2*i*width);

      std::reverse_copy(sig[i].begin(), sig[i].end, 
                        host_signals+(2*i+1)*width+(m-(int)sig[i].size()));
    }

    Decimal *device_signals;
    hipMalloc(&device_signals, size_bytes);
    hipMemcpy(device_signals, host_signals, size_bytes, 
               hipMemcpyHostToDevice);

    // 2. init result data

    Decimal *device_param;
    int *device_wsize;

    hipMalloc(&device_param, size_bytes);
    hipMalloc(&device_param, 2*n*sizeof(int));

    // 3. init kernel

    // 4. start kernel

    // 5. retrieve and post-calculate results

    Decimal *host_param = new Decimal[2*n*m];
    int *host_wsize = new int[2*n*m];

    hipMemcpy(host_param, device_param, size_bytes, 
               hipMemcpyDeviceToHost);

    hipMemcpy(host_wsize, device_wsize, 2*n*sizeof(int), 
               hipMemcpyDeviceToHost);

    res.resize(n);
    for(int i = 0; i < n; ++i) {
      Decimal pvleft, pvright;  // param value
      int wsleft, wsright;  // window size

      for(int pos = 0; pos < (int)sig[i].size(); ++pos) {
        pvright = host_param[2*i*m + pos];
        pvleft = host_param[(2*i+2)*m - (pos+1)]; // (2*i+1)*m + (m-pos-1)
        wsright = host_wsize[2*i*m + pos];
        wsleft = host_wsize[(2*i+2)*m - (pos+1)];
        
        res.back()->addLeft(wsleft);
        res.back()->addRight(wsright);
        res.back()->addDenoised((pvright * wsright + pvleft * wsleft) /
                                (wsleft + wsright));
      }
    }

    // 6. clean up

    delete[] host_signals;
    delete[] host_param;
    delete[] host_wsize;

    hipFree(device_signals);
    hipFree(device_param);
    hipFree(device_wsize);
  }
                                      
}  // namespace

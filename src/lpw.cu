#include "hip/hip_runtime.h"
/*
  Copyright: (c) 2011 Matija Osrecki <matija.osrecki@fer.hr>
 */

#include <cassert>
#include <cstdio>
#include <vector>

#include "lpw.h"

using std::vector;

/*
  Copies vector to regular array.
*/
float* copy_vector(const vecd &v) {
  int n = (int)v.size();
  float *va = new float[n];

  for(int i = 0; i < n; ++i) 
    va[i] = (float)v[i];

  return va;
}

/*
  Takes two arrays of sizes N, and calculates for every i = [1,n]:
    result[i] = result[i-1] + a1[i-1] * a2[a-1], result[0] = 0

  The resulting array is accumulated sum of multiplied array a1 and a2.
*/
float* multiply(const float *a1, const float *a2, const int size) {
  float *res = new float[size+1];

  res[0] = 0.0f;
  for(int i = 0; i < size; ++i) 
    res[i+1] = res[i] + a1[i] * a2[i];

  return res;
}

/*
  Allocates memory of size 4 * n bytes on device and copies input array to device.
*/
float* alloc_cpy(const float *src, const int n, const hipMemcpyKind type) {
  float *dest;

  hipMalloc(&dest, n * sizeof(float));
  hipMemcpy(dest, src, n * sizeof(float), type);

  return dest;
}


/*
  Function for device excecution. 
  Calculates B and E matrix members.
 */
__global__ void lsw_kernel(const float *dx, const float *dy,
			   const float *dxx, const float *dxy,
			   float *db, float *derr,
			   const int n, const int tspread) {

  int wsize = blockIdx.y * blockDim.y + threadIdx.y;
  int wpos = blockIdx.x * blockDim.x * tspread + threadIdx.x;

  if(wsize >= n-1) return;

  for(int i = 0; i < tspread; ++i) {
    if(wpos + wsize + 2 >= n+1) return;
    
    float b = (dxy[wpos + wsize + 2] - dxy[wpos]) / (dxx[wpos + wsize + 2] - dxx[wpos]);
    float err = 0.0f, delta;

    for(int j = wpos; j < wpos + wsize + 2; ++j) {
      delta = dy[j] - b * dx[j];
      err += delta * delta;
    }

    db[wsize * (n-1) + wpos] = b;
    derr[wsize * (n-1) + wpos] = err;

    wpos += blockDim.x;
  }
}

void lsw(const vecd &y, const vecd &x, lpw_res &result) {
  assert(x.size() == y.size());
  
  int n = (int)x.size();

  // copy data from vectors
  float *hy = copy_vector(y);
  float *hx = copy_vector(x);

  // calculate Xt * X and Xt * Y
  float *hxx = multiply(hx, hx, n);
  float *hxy = multiply(hx, hy, n);

  // allocate device memory and copy data
  float *dy = alloc_cpy(hy, n, hipMemcpyHostToDevice);
  float *dx = alloc_cpy(hx, n, hipMemcpyHostToDevice);
  float *dxx = alloc_cpy(hxx, n+1, hipMemcpyHostToDevice);
  float *dxy = alloc_cpy(hxy, n+1, hipMemcpyHostToDevice);

  // allocate device memory for resulting B matrix
  float *db, *derr;
  
  int width = (n-1) * sizeof(float);

  hipMalloc(&db, width * (n-1));
  hipMalloc(&derr, width *(n-1));

  // init kernel
  const int block_size = 16;
  const int thread_spread = 64;

  const int grid_width = (n-2) / (block_size*thread_spread) + 1;
  const int grid_height = (n-2) / block_size + 1;

  dim3 dim_block(block_size, block_size);
  dim3 dim_grid(grid_width, grid_height); 

  // start kernel that computes B matrix
  lsw_kernel<<<dim_grid, dim_block>>>(dx, dy, dxx, dxy, db, derr, n, thread_spread);

  // retrieve results
  float *hb = new float[(n-1) * (n-1)];
  hipMemcpy(hb, db, width * (n-1), hipMemcpyDeviceToHost);

  float *herr = new float[(n-1) * (n-1)];
  hipMemcpy(herr, derr, width * (n-1), hipMemcpyDeviceToHost);

  // store results to vector
  result.resize(n-1, std::vector<pdd> (n-1));
  for(int wsize = 0; wsize < n-1; ++wsize)
    for(int wpos = 0; wpos < n-1; ++wpos) {
      result[wsize][wpos].first = hb[wsize * (n-1) + wpos];
      result[wsize][wpos].second = herr[wsize * (n-1) + wpos];
    }

  // free memory
  delete[] hx;
  delete[] hy;
  delete[] hxx;
  delete[] hxy;
  delete[] hb;
  delete[] herr;

  hipFree(dx);
  hipFree(dy);
  hipFree(dxx);
  hipFree(dxy);
  hipFree(db);
  hipFree(derr);
}

